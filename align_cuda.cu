#include "hip/hip_runtime.h"
/*
 * Exact genetic sequence alignment
 * (Using brute force)
 *
 * CUDA version
 *
 * Computacion Paralela, Grado en Informatica (Universidad de Valladolid)
 * 2023/2024
 *
 * v1.3
 *
 * (c) 2024, Arturo Gonzalez-Escribano
 */
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include<limits.h>
#include<sys/time.h>

/* Headers for the CUDA+MPI assignment versions */
#include<hip/hip_runtime.h>
#include<mpi.h>


/* Example of macros for error checking in CUDA */
#define CUDA_CHECK_FUNCTION( call )	{ hipError_t check = call; if ( check != hipSuccess ) fprintf(stderr, "CUDA Error in line: %d, %s\n", __LINE__, hipGetErrorString(check) ); }
#define CUDA_CHECK_KERNEL( )	{ hipError_t check = hipGetLastError(); if ( check != hipSuccess ) fprintf(stderr, "CUDA Kernel Error in line: %d, %s\n", __LINE__, hipGetErrorString(check) ); }

/* Arbitrary value to indicate that no matches are found */
#define	NOT_FOUND	-1
#define NOT_FOUND_ULL 18446744073709551615ULL

/* Arbitrary value to restrict the checksums period */
#define CHECKSUM_MAX	65535

/* 
 * Utils: Function to get wall time
 */
double cp_Wtime(){
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec + 1.0e-6 * tv.tv_usec;
}

/*
 * Utils: Random generator
 */
#include "rng.c"



/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */
/* ADD KERNELS AND OTHER FUNCTIONS HERE */


/**	Kernel CUDA per trovare la prima occorrenza di ogni pattern in una sequenza
 * 
 * Input:	
 * 	sequence: 		Sequenza di caratteri da analizzare
 * 	pattern_buffer: Buffer continuo contenente tutti i pattern
 *  pat_offsets: 	Offset di ciascun pattern nel buffer
 * 	pat_length: 	Lunghezza di ciascun pattern
 * 	pat_found: 		Array per memorizzare la posizione minima trovata per ogni pattern
 * 	pat_number: 	Numero totale di pattern
 *  seq_length: 	Lunghezza della sequenza
 * 	rank: 			Rank del processo MPI
 * 	my_seq_length:	Lunghezza della porzione di sequenza per processo
 * */ 
__global__ void findPatterns(char *sequence, char *pattern_buffer, int *pat_offsets, unsigned long *pat_length, unsigned long long *pat_found, 
                             int pat_number, unsigned long seq_length, int rank, unsigned long my_seq_length) 
{
    extern __shared__ unsigned long long shared_min[];	// Shared memory per memorizzare il minimo locale del blocco

    const int pattern_idx = blockIdx.x;	// Indice del pattern (un pattern per blocco)
    const int local_start = threadIdx.x + blockIdx.y * blockDim.x;	// Posizione iniziale locale nel blocco
    const int global_start = local_start + (rank * my_seq_length);	// Posizione globale nella sequenza (aggiustata per il rank MPI)

    // --- Inizializzazione della shared memory ---
    if (threadIdx.x == 0) {
        shared_min[0] = (unsigned long long)NOT_FOUND;	// Il thread 0 del blocco inizializza il minimo locale a NOT_FOUND
    }
    __syncthreads(); // Sincronizza i thread *del blocco*

    // --- Controllo di validità ---
    if (pattern_idx >= pat_number || global_start + pat_length[pattern_idx] > seq_length) {
        return;	// Esce se il pattern non esiste o se la posizione supera la sequenza
    }

    // --- Confronto del pattern ---
    const int pattern_start = pat_offsets[pattern_idx];	// Offset del pattern corrente nel buffer continuo
    bool is_match = true; // Flag per indicare un match valido
    for (int i = 0; i < pat_length[pattern_idx]; i++) {
        if (sequence[global_start + i] != pattern_buffer[pattern_start + i]) {  // Confronta carattere per carattere
            is_match = false;
            break;
        }
    }

    // --- Aggiornamento del minimo locale ---
    // Se c'è un match, aggiorna il minimo nella shared memory
    if (is_match && (shared_min[0] == (unsigned long long)NOT_FOUND || global_start < shared_min[0])) {
        // Compare and Swap, serve per passare da uno stato indefinito (NOT_FOUND) a un valore valido senza conflitti. 
		// Permette di sapere se sei il primo thread a scrivere (tramite il valore restituito).
        unsigned long long old_min = atomicCAS(&shared_min[0], (unsigned long long)NOT_FOUND, (unsigned long long)global_start);
        // Serve per competere tra più thread che hanno valori validi, mantenendo solo il più piccolo
        if (old_min != (unsigned long long)NOT_FOUND && global_start < old_min) {
            atomicMin(&shared_min[0], (unsigned long long)global_start);
        }
    }
    __syncthreads(); // Sincronizza prima dell'aggiornamento globale

    // --- Scrittura del risultato globale ---
    // Solo il thread 0 del blocco aggiorna pat_found con il minimo locale
    if (threadIdx.x == 0 && shared_min[0] != (unsigned long long)NOT_FOUND) {
        unsigned long long old_found = atomicCAS(&pat_found[pattern_idx], (unsigned long long)NOT_FOUND, shared_min[0]); // Usa atomicCAS per inizializzare pat_found
        if (old_found != (unsigned long long)NOT_FOUND) {
            atomicMin(&pat_found[pattern_idx], shared_min[0]);	// Se già inizializzato, aggiorna con il valore minimo
        }
    }
}





/**	Kernel CUDA per aggiornare i contatori di match nella sequenza
 *  
 * Input:
 * 	pat_found: Posizioni minime dei pattern trovati (o NOT_FOUND_ULL se non trovati)
 * 	pat_length: Lunghezza di ciascun pattern
 * 	seq_matches: Array dei contatori di match per ogni posizione della sequenza
 * 	pat_number: Numero di pattern locali gestiti dal processo
 * 	seq_length: Lunghezza totale della sequenza
 * */ 
__global__ void updateSeqMatches(unsigned long long *pat_found, unsigned long *pat_length, int *seq_matches, int pat_number, unsigned long seq_length) 
{
    // Calcola l'indice del pattern per questo thread
    const int pattern_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Controllo di validità: esce se il pattern non esiste
    if (pattern_idx >= pat_number) {
        return;
    }

    // Ottiene la posizione del match per il pattern
    const unsigned long long match_pos = pat_found[pattern_idx];
    // Esce se il pattern non è stato trovato
    if (match_pos == NOT_FOUND_ULL) {
        return;
    }

    // Aggiorna i contatori di seq_matches per ogni posizione coperta dal pattern
    for (int i = 0; i < pat_length[pattern_idx]; i++) {
        // Incrementa atomicamente il contatore nella posizione match_pos + i
        atomicAdd(&seq_matches[match_pos + i], 1);
    }
}










/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

/*
 * Function: Allocate new patttern
 */
char *pattern_allocate( rng_t *random, unsigned long pat_rng_length_mean, unsigned long pat_rng_length_dev, unsigned long seq_length, unsigned long *new_length ) {

	/* Random length */
	unsigned long length = (unsigned long)rng_next_normal( random, (double)pat_rng_length_mean, (double)pat_rng_length_dev );
	if ( length > seq_length ) length = seq_length;
	if ( length <= 0 ) length = 1;

	/* Allocate pattern */
	char *pattern = (char *)malloc( sizeof(char) * length );
	if ( pattern == NULL ) {
		fprintf(stderr,"\n-- Error allocating a pattern of size: %lu\n", length );
		exit( EXIT_FAILURE );
	}

	/* Return results */
	*new_length = length;
	return pattern;
}

/*
 * Function: Fill random sequence or pattern
 */
void generate_rng_sequence( rng_t *random, float prob_G, float prob_C, float prob_A, char *seq, unsigned long length) {
	unsigned long ind; 
	for( ind=0; ind<length; ind++ ) {
		double prob = rng_next( random );
		if( prob < prob_G ) seq[ind] = 'G';
		else if( prob < prob_C ) seq[ind] = 'C';
		else if( prob < prob_A ) seq[ind] = 'A';
		else seq[ind] = 'T';
	}
}

/*
 * Function: Copy a sample of the sequence
 */
void copy_sample_sequence( rng_t *random, char *sequence, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length) {
	/* Choose location */
	unsigned long  location = (unsigned long)rng_next_normal( random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev );
	if ( location > seq_length - length ) location = seq_length - length;
	if ( location <= 0 ) location = 0;

	/* Copy sample */
	unsigned long ind; 
	for( ind=0; ind<length; ind++ )
		pattern[ind] = sequence[ind+location];
}

/*
 * Function: Regenerate a sample of the sequence
 */
void generate_sample_sequence( rng_t *random, rng_t random_seq, float prob_G, float prob_C, float prob_A, unsigned long seq_length, unsigned long pat_samp_loc_mean, unsigned long pat_samp_loc_dev, char *pattern, unsigned long length ) {
	/* Choose location */
	unsigned long  location = (unsigned long)rng_next_normal( random, (double)pat_samp_loc_mean, (double)pat_samp_loc_dev );
	if ( location > seq_length - length ) location = seq_length - length;
	if ( location <= 0 ) location = 0;

	/* Regenerate sample */
	rng_t local_random = random_seq;
	rng_skip( &local_random, location );
	generate_rng_sequence( &local_random, prob_G, prob_C, prob_A, pattern, length);
}


/*
 * Function: Print usage line in stderr
 */
void show_usage( char *program_name ) {
	fprintf(stderr,"Usage: %s ", program_name );
	fprintf(stderr,"<seq_length> <prob_G> <prob_C> <prob_A> <pat_rng_num> <pat_rng_length_mean> <pat_rng_length_dev> <pat_samples_num> <pat_samp_length_mean> <pat_samp_length_dev> <pat_samp_loc_mean> <pat_samp_loc_dev> <pat_samp_mix:B[efore]|A[fter]|M[ixed]> <long_seed>\n");
	fprintf(stderr,"\n");
}



/*
 * MAIN PROGRAM
 */
int main(int argc, char *argv[]) {
	/* 0. Disattiva il buffering per stdout e stderr garantendo che i messaggi di debug o errore vengano scritti immediatamente	*/
	setbuf(stdout, NULL);
	setbuf(stderr, NULL);

	/* 1. Read scenary arguments */
	
	/* 1.0. Init MPI before processing arguments */
	MPI_Init(&argc, &argv);	// Passare &argc, &argv è una buona pratica perché permette a MPI di gestire eventuali opzioni della riga di comando
	int rank, size;
	MPI_Comm_rank( MPI_COMM_WORLD, &rank );
	MPI_Comm_size(MPI_COMM_WORLD, &size);

	/* 1.1. Check minimum number of arguments */
	if (argc < 15) {
		fprintf(stderr, "\n-- Error: Not enough arguments when reading configuration from the command line\n\n");
		show_usage( argv[0] );
		MPI_Abort( MPI_COMM_WORLD, EXIT_FAILURE );
	}

	/* 1.2. Read argument values */
	unsigned long seq_length = atol( argv[1] );
	float prob_G = atof( argv[2] );
	float prob_C = atof( argv[3] );
	float prob_A = atof( argv[4] );
	if ( prob_G + prob_C + prob_A > 1 ) {
		fprintf(stderr, "\n-- Error: The sum of G,C,A,T nucleotid probabilities cannot be higher than 1\n\n");
		show_usage( argv[0] );
		MPI_Abort( MPI_COMM_WORLD, EXIT_FAILURE );
	}
	prob_C += prob_G;
	prob_A += prob_C;

	int pat_rng_num = atoi( argv[5] );
	unsigned long pat_rng_length_mean = atol( argv[6] );
	unsigned long pat_rng_length_dev = atol( argv[7] );
	
	int pat_samp_num = atoi( argv[8] );
	unsigned long pat_samp_length_mean = atol( argv[9] );
	unsigned long pat_samp_length_dev = atol( argv[10] );
	unsigned long pat_samp_loc_mean = atol( argv[11] );
	unsigned long pat_samp_loc_dev = atol( argv[12] );

	char pat_samp_mix = argv[13][0];
	if ( pat_samp_mix != 'B' && pat_samp_mix != 'A' && pat_samp_mix != 'M' ) {
		fprintf(stderr, "\n-- Error: Incorrect first character of pat_samp_mix: %c\n\n", pat_samp_mix);
		show_usage( argv[0] );
		MPI_Abort( MPI_COMM_WORLD, EXIT_FAILURE );
	}

	unsigned long seed = atol( argv[14] );

#ifdef DEBUG
	/* DEBUG: Print arguments */
	printf("\nArguments: seq_length=%lu\n", seq_length );
	printf("Arguments: Accumulated probabilitiy G=%f, C=%f, A=%f, T=1\n", prob_G, prob_C, prob_A );
	printf("Arguments: Random patterns number=%d, length_mean=%lu, length_dev=%lu\n", pat_rng_num, pat_rng_length_mean, pat_rng_length_dev );
	printf("Arguments: Sample patterns number=%d, length_mean=%lu, length_dev=%lu, loc_mean=%lu, loc_dev=%lu\n", pat_samp_num, pat_samp_length_mean, pat_samp_length_dev, pat_samp_loc_mean, pat_samp_loc_dev );
	printf("Arguments: Type of mix: %c, Random seed: %lu\n", pat_samp_mix, seed );
	printf("\n");
#endif // DEBUG


	int local_rank;		// Rank locale all'interno del nodo
	MPI_Comm local_comm;	// Comunicatore per i processi che condividono lo stesso nodo

	// Suddivisione di MPI_COMM_WORLD in sotto-comunicatori, uno per ogni nodo fisico
	// MPI_COMM_TYPE_SHARED raggruppa i processi MPI che condividono la stessa memoria (quindi lo stesso nodo)
	// Il valore `rank` è usato come "chiave" per mantenere l'ordine originale dei rank
	// MPI_INFO_NULL indica che non vengono passate informazioni aggiuntive
	MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL, &local_comm);
	MPI_Comm_rank(local_comm, &local_rank);	// Ottenimento del rank del processo all'interno del nodo locale

	int deviceCount;
	CUDA_CHECK_FUNCTION(hipGetDeviceCount(&deviceCount));	// Numero di GPU disponibili sul nodo
	int gpu_id = local_rank % deviceCount; 	// Ogni processo MPI all'interno del nodo ottiene una GPU in modo ciclico usando il modulo (%)
	CUDA_CHECK_FUNCTION(hipSetDevice(gpu_id));	// Impostazione della GPU da usare per questo processo MPI

	// DEBUG
	int dev_id;
	CUDA_CHECK_FUNCTION(hipGetDevice(&dev_id));
	printf("MPI Rank %d (Local Rank %d) usa GPU %d\n", rank, local_rank, dev_id);


	/* 2. Initialize data structures */
	/* 2.1. Skip allocate and fill sequence */
	rng_t random = rng_new( seed );
	rng_skip( &random, seq_length );

	/* 2.2. Allocate and fill patterns */
	/* 2.2.1 Allocate main structures */
	int pat_number = pat_rng_num + pat_samp_num;
	unsigned long *pat_length = (unsigned long *)malloc( sizeof(unsigned long) * pat_number );
	char **pattern = (char **)malloc( sizeof(char*) * pat_number );
	if ( pattern == NULL || pat_length == NULL ) {
		fprintf(stderr,"\n-- Error allocating the basic patterns structures for size: %d\n", pat_number );
		MPI_Abort( MPI_COMM_WORLD, EXIT_FAILURE );
	}

	/* 2.2.2 Allocate and initialize ancillary structure for pattern types */
	int ind;
	unsigned long lind;
	#define PAT_TYPE_NONE	0
	#define PAT_TYPE_RNG	1
	#define PAT_TYPE_SAMP	2
	char *pat_type = (char *)malloc( sizeof(char) * pat_number );
	if ( pat_type == NULL ) {
		fprintf(stderr,"\n-- Error allocating ancillary structure for pattern of size: %d\n", pat_number );
		MPI_Abort( MPI_COMM_WORLD, EXIT_FAILURE );
	}
	for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_NONE;

	/* 2.2.3 Fill up pattern types using the chosen mode */
	switch( pat_samp_mix ) {
	case 'A':
		for( ind=0; ind<pat_rng_num; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		for( ; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		break;
	case 'B':
		for( ind=0; ind<pat_samp_num; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		for( ; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		break;
	default:
		if ( pat_rng_num == 0 ) {
			for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_SAMP;
		}
		else if ( pat_samp_num == 0 ) {
			for( ind=0; ind<pat_number; ind++ ) pat_type[ind] = PAT_TYPE_RNG;
		}
		else if ( pat_rng_num < pat_samp_num ) {
			int interval = pat_number / pat_rng_num;
			for( ind=0; ind<pat_number; ind++ ) 
				if ( (ind+1) % interval == 0 ) pat_type[ind] = PAT_TYPE_RNG;
				else pat_type[ind] = PAT_TYPE_SAMP;
		}
		else {
			int interval = pat_number / pat_samp_num;
			for( ind=0; ind<pat_number; ind++ ) 
				if ( (ind+1) % interval == 0 ) pat_type[ind] = PAT_TYPE_SAMP;
				else pat_type[ind] = PAT_TYPE_RNG;
		}
	}

	/* 2.2.4 Generate the patterns */
	for( ind=0; ind<pat_number; ind++ ) {
		if ( pat_type[ind] == PAT_TYPE_RNG ) {
			pattern[ind] = pattern_allocate( &random, pat_rng_length_mean, pat_rng_length_dev, seq_length, &pat_length[ind] );
			generate_rng_sequence( &random, prob_G, prob_C, prob_A, pattern[ind], pat_length[ind] );
		}
		else if ( pat_type[ind] == PAT_TYPE_SAMP ) {
			pattern[ind] = pattern_allocate( &random, pat_samp_length_mean, pat_samp_length_dev, seq_length, &pat_length[ind] );
#define REGENERATE_SAMPLE_PATTERNS
#ifdef REGENERATE_SAMPLE_PATTERNS
			rng_t random_seq_orig = rng_new( seed );
			generate_sample_sequence( &random, random_seq_orig, prob_G, prob_C, prob_A, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#else
			copy_sample_sequence( &random, sequence, seq_length, pat_samp_loc_mean, pat_samp_loc_dev, pattern[ind], pat_length[ind] );
#endif
		}
		else {
			fprintf(stderr,"\n-- Error internal: Paranoic check! A pattern without type at position %d\n", ind );
			MPI_Abort( MPI_COMM_WORLD, EXIT_FAILURE );
		}
	}
	free( pat_type );

	/* Allocate and move the patterns to the GPU */
	unsigned long *d_pat_length;
	char **d_pattern;
	CUDA_CHECK_FUNCTION( hipMalloc( &d_pat_length, sizeof(unsigned long) * pat_number ) );	// Array che memorizza la lunghezza di ciascun pattern
	CUDA_CHECK_FUNCTION( hipMalloc( &d_pattern, sizeof(char *) * pat_number ) );	// Array che punta a ciascun pattern

	// Array di puntatori a char * allocato sulla memoria host che fungerà da buffer per memorizzare i pattern prima di trasferirli sulla GPU
	char **d_pattern_in_host = (char **)malloc( sizeof(char*) * pat_number );
	if ( d_pattern_in_host == NULL ) {
		fprintf(stderr,"\n-- Error allocating the patterns structures replicated in the host for size: %d\n", pat_number );
		MPI_Abort( MPI_COMM_WORLD, EXIT_FAILURE );
	}

	/*
	* Per ogni pattern il codice:
	*     Alloca memoria sulla GPU per ciascun pattern (memoria per ogni singolo pattern di lunghezza pat_length[ind]).
    *     Copia il pattern dalla memoria host alla memoria device utilizzando hipMemcpy, passando i dati da pattern[ind] (memoria host) 
	* 		a d_pattern_in_host[ind] (memoria device)
	*/
	for (ind=0; ind<pat_number; ind++) {
		CUDA_CHECK_FUNCTION(hipMalloc(&(d_pattern_in_host[ind]), sizeof(char *) * pat_length[ind]));
        CUDA_CHECK_FUNCTION(hipMemcpy(d_pattern_in_host[ind], pattern[ind], pat_length[ind] * sizeof(char), hipMemcpyHostToDevice));
	}

	/*
	* Una volta che tutti i pattern sono stati allocati e copiati nella memoria device, 
	* copia l'array di puntatori d_pattern_in_host (memoria host) nell'array di puntatori d_pattern (memoria device), 
	* permettendo al codice CUDA di accedere ai pattern sulla GPU
	*/
	CUDA_CHECK_FUNCTION( hipMemcpy( d_pattern, d_pattern_in_host, pat_number * sizeof(char *), hipMemcpyHostToDevice ) );

	/* Avoid the usage of arguments to take strategic decisions
	 * In a real case the user only has the patterns and sequence data to analize
	 */
	argc = 0;
	argv = NULL;
	pat_rng_num = 0;
	pat_rng_length_mean = 0;
	pat_rng_length_dev = 0;
	pat_samp_num = 0;
	pat_samp_length_mean = 0;
	pat_samp_length_dev = 0;
	pat_samp_loc_mean = 0;
	pat_samp_loc_dev = 0;
	pat_samp_mix = '0';

	/* 2.3. Other result data and structures */
	int pat_matches = 0;

	/* 2.3.1. Other results related to patterns */
	unsigned long long *pat_found;
	pat_found = (unsigned long long*)malloc( sizeof(unsigned long long) * pat_number );
	if ( pat_found == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux pattern structure for size: %d\n", pat_number );
		MPI_Abort( MPI_COMM_WORLD, EXIT_FAILURE );
	}
	
	/* 3. Start global timer */
	// Le barriere servono poichè potrebbe esserci un rischio che il timer venga avviato prima che tutte le operazioni siano effettivamente completate, distorcendo la misurazione del tempo */
    CUDA_CHECK_FUNCTION( hipDeviceSynchronize() );	// Evita lavoro GPU pendente prima di sincronizzarsi con MPI
	MPI_Barrier( MPI_COMM_WORLD );	// Tutti i processi si allineano prima di partire
	double ttotal = cp_Wtime();

/*
 *
 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
 * DO NOT USE OpenMP IN YOUR CODE
 *
 */
	/* 2.1. Allocate and fill sequence */
	char *sequence = (char *)malloc( sizeof(char) * seq_length );
	if ( sequence == NULL ) {
		fprintf(stderr,"\n-- Error allocating the sequence for size: %lu\n", seq_length );
		MPI_Abort( MPI_COMM_WORLD, EXIT_FAILURE );
	}

	random = rng_new( seed );
	generate_rng_sequence( &random, prob_G, prob_C, prob_A, sequence, seq_length);

#ifdef DEBUG
	/* DEBUG: Print sequence and patterns */
	printf("-----------------\n");
	printf("Sequence: ");
	for( lind=0; lind<seq_length; lind++ ) 
		printf( "%c", sequence[lind] );
	printf("\n-----------------\n");
	printf("Patterns: %d ( rng: %d, samples: %d )\n", pat_number, pat_rng_num, pat_samp_num );
	int debug_pat;
	for( debug_pat=0; debug_pat<pat_number; debug_pat++ ) {
		printf( "Pat[%d]: ", debug_pat );
		for( lind=0; lind<pat_length[debug_pat]; lind++ ) 
			printf( "%c", pattern[debug_pat][lind] );
		printf("\n");
	}
	printf("-----------------\n\n");
#endif // DEBUG

	/* 2.3.2. Other results related to the main sequence */
	int *seq_matches;
	seq_matches = (int *)malloc( sizeof(int) * seq_length );
	if ( seq_matches == NULL ) {
		fprintf(stderr,"\n-- Error allocating aux sequence structures for size: %lu\n", seq_length );
		MPI_Abort( MPI_COMM_WORLD, EXIT_FAILURE );
	}


	/* 4. Initialize ancillary structures */
	for( ind=0; ind<pat_number; ind++) {
		pat_found[ind] = (unsigned long long)NOT_FOUND;
	}
	for( lind=0; lind<seq_length; lind++) {
		seq_matches[lind] = NOT_FOUND;
	}



	/* 5. Search for each pattern */


	// --- Preparazione del buffer concatenato per i pattern ---

	// Calcola la lunghezza totale di tutti i pattern
	int total_pattern_length = 0;
	for (int i = 0; i < pat_number; i++) {
		total_pattern_length += pat_length[i];
	}

	// Alloca memoria sull'host per il buffer concatenato e gli offset
	char *h_pattern_buffer = (char*)malloc(total_pattern_length * sizeof(char));
	int *h_pattern_offsets = (int*)malloc(pat_number * sizeof(int));
	if (!h_pattern_buffer || !h_pattern_offsets) {
		fprintf(stderr, "Errore: Allocazione memoria host fallita\n");
		MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
	}

	// Riempie il buffer concatenato con i pattern e calcola gli offset
	int current_offset = 0;
	for (int i = 0; i < pat_number; i++) {
		memcpy(h_pattern_buffer + current_offset, pattern[i], pat_length[i] * sizeof(char));
		h_pattern_offsets[i] = current_offset;
		current_offset += pat_length[i];
	}

	// --- Allocazione memoria sulla GPU ---

	// Alloca buffer per i pattern e gli offset sul device
	char *d_pattern_buffer;
	int *d_pattern_offsets;
	CUDA_CHECK_FUNCTION(hipMalloc((void**)&d_pattern_buffer, total_pattern_length * sizeof(char)));
	CUDA_CHECK_FUNCTION(hipMalloc((void**)&d_pattern_offsets, pat_number * sizeof(int)));

	// --- Copia dei dati dalla CPU alla GPU ---

	CUDA_CHECK_FUNCTION(hipMemcpy(d_pattern_buffer, h_pattern_buffer, total_pattern_length * sizeof(char), hipMemcpyHostToDevice));
	CUDA_CHECK_FUNCTION(hipMemcpy(d_pattern_offsets, h_pattern_offsets, pat_number * sizeof(int), hipMemcpyHostToDevice));

	// --- Allocazione memoria per il kernel findPatterns ---

	// Allocazione array sul device
	char *d_sequence;               // Sequenza di caratteri
	int *d_seq_matches;             // Contatore di match per posizione
	unsigned long long *d_pat_found; // Posizioni minime dei pattern
	CUDA_CHECK_FUNCTION(hipMalloc((void**)&d_sequence, seq_length * sizeof(char)));
	CUDA_CHECK_FUNCTION(hipMalloc((void**)&d_pat_found, pat_number * sizeof(unsigned long long)));
	CUDA_CHECK_FUNCTION(hipMalloc((void**)&d_seq_matches, seq_length * sizeof(int)));

	// Allocazione array sull'host
	unsigned long long *h_pat_found = (unsigned long long*)malloc(pat_number * sizeof(unsigned long long));
	int *h_seq_matches = NULL;
	if (rank == 0) {
		h_seq_matches = (int*)malloc(seq_length * sizeof(int));
	}
	if (!h_pat_found || (rank == 0 && !h_seq_matches)) {
		fprintf(stderr, "Errore: Allocazione memoria host fallita\n");
		MPI_Abort(MPI_COMM_WORLD, EXIT_FAILURE);
	}

	// --- Inizializzazione memoria sul device ---

	CUDA_CHECK_FUNCTION(hipMemcpy(d_sequence, sequence, seq_length * sizeof(char), hipMemcpyHostToDevice));	// Copia la sequenza dall'host al device
	CUDA_CHECK_FUNCTION(hipMemset(d_pat_found, NOT_FOUND, pat_number * sizeof(unsigned long long)));	// Inizializza d_pat_found con NOT_FOUND
	CUDA_CHECK_FUNCTION(hipMemcpy(d_pat_length, pat_length, pat_number * sizeof(unsigned long), hipMemcpyHostToDevice)); // Copia le lunghezze dei pattern dall'host al device

	// Inizializzazione d_seq_matches: -1 per rank 0, 0 per altri rank
	// Nota: rank 0 usa -1 per evitare conteggi doppi dei match nelle porzioni della sequenza
	if (rank == 0) {
		CUDA_CHECK_FUNCTION(hipMemset(d_seq_matches, -1, seq_length * sizeof(int)));
	} else {
		CUDA_CHECK_FUNCTION(hipMemset(d_seq_matches, 0, seq_length * sizeof(int)));
	}

	// --- Configurazione e lancio del kernel findPatterns ---

	// Parametri del kernel
	const dim3 blockSize(128); // Thread per blocco 
	const unsigned long my_seq_length = seq_length / size; // Porzione di sequenza per processo
	const size_t maxGridY = 65535; // Limite massimo CUDA per dimensione griglia Y
	size_t gridY = (my_seq_length + blockSize.x - 1) / blockSize.x; // Blocchi necessari per coprire my_seq_length
	gridY = (gridY > maxGridY) ? maxGridY : gridY; // Rispetta il limite CUDA
	const size_t sharedMemSize = sizeof(unsigned long long); // 8 byte per il minimo locale

	// Lancia il kernel findPatterns
	findPatterns<<<dim3(pat_number, gridY), blockSize, sharedMemSize>>>(
		d_sequence, d_pattern_buffer, d_pattern_offsets, d_pat_length, d_pat_found, pat_number, seq_length, rank, my_seq_length);

	// --- Post-elaborazione dei risultati ---

	// Copia i risultati di d_pat_found sull'host (l'operazione non comincia fino al completamento del kernel poiché riconosce il trasferimento di dati)
	CUDA_CHECK_FUNCTION(hipMemcpy(pat_found, d_pat_found, pat_number * sizeof(unsigned long long), hipMemcpyDeviceToHost));

	// Riduce i minimi locali di tutti i processi per ottenere il minimo globale
	MPI_Allreduce(pat_found, h_pat_found, pat_number, MPI_UNSIGNED_LONG_LONG, MPI_MIN, MPI_COMM_WORLD);

	// Conta il numero di pattern trovati (solo su CPU per semplicità)
	for (int j = 0; j < pat_number; j++) {
		if (h_pat_found[j] != NOT_FOUND_ULL) {
			pat_matches++;
		}
	}




	// --- Configurazione del secondo kernel: updateSeqMatches --- 

	const int local_pat_number = (rank < pat_number % size) ? (pat_number / size + 1) : (pat_number / size); // Calcola il numero di pattern assegnati a questo processo MPI 
	const int start_pat = (rank < pat_number % size) ? (rank * local_pat_number) : (rank * (pat_number / size) + pat_number % size); // Calcola l'indice del primo pattern assegnato a questo processo

	// Alloca un array sul device per la porzione locale di h_pat_found (necessario perché d_pat_found contiene solo i dati locali del processo)
	unsigned long long *d_pat_found_split;
	CUDA_CHECK_FUNCTION(hipMalloc((void**)&d_pat_found_split, local_pat_number * sizeof(unsigned long long)));
	CUDA_CHECK_FUNCTION(hipMemcpy(d_pat_found_split, &h_pat_found[start_pat], local_pat_number * sizeof(unsigned long long), hipMemcpyHostToDevice));

	// Configura i parametri del kernel 
	const dim3 blockSize2(256); // Numero di thread per blocco
	const dim3 gridSize2((local_pat_number + blockSize2.x - 1) / blockSize2.x); // Numero di blocchi per coprire tutti i pattern locali. Griglia 1D. 

	// Lancia il kernel updateSeqMatches per aggiornare d_seq_matches 
	updateSeqMatches<<<gridSize2, blockSize2>>>(d_pat_found_split, &d_pat_length[start_pat], d_seq_matches, local_pat_number, seq_length);

	// --- Post-elaborazione dei risultati --- 

	// Copia d_seq_matches dall'host alla CPU. Nota: hipMemcpy blocca, quindi non serve hipDeviceSynchronize 
	CUDA_CHECK_FUNCTION(hipMemcpy(seq_matches, d_seq_matches, seq_length * sizeof(int), hipMemcpyDeviceToHost));

	// Riduce i contatori locali di tutti i processi in h_seq_matches (solo rank 0) 
	MPI_Reduce(seq_matches, h_seq_matches, seq_length, MPI_INT, MPI_SUM, 0, MPI_COMM_WORLD);

	// --- Deallocazione memoria --- 

	// Libera gli array allocati sul device 
	CUDA_CHECK_FUNCTION(hipFree(d_sequence));
	CUDA_CHECK_FUNCTION(hipFree(d_pat_found));
	CUDA_CHECK_FUNCTION(hipFree(d_seq_matches));
	CUDA_CHECK_FUNCTION(hipFree(d_pat_found_split));
	CUDA_CHECK_FUNCTION(hipFree(d_pat_length));
	CUDA_CHECK_FUNCTION(hipFree(d_pattern_buffer));
	CUDA_CHECK_FUNCTION(hipFree(d_pattern_offsets));

	// Libera l'array di puntatori ai pattern sul device 
	for (int ind = 0; ind < pat_number; ind++) {
		CUDA_CHECK_FUNCTION(hipFree(d_pattern_in_host[ind]));
	}
	CUDA_CHECK_FUNCTION(hipFree(d_pattern));

	// Libera i buffer allocati sull'host 
	free(d_pattern_in_host);
	free(h_pattern_buffer);
	free(h_pattern_offsets);


	/* 7. Calcolo dei checksum e deallocazione memoria finale */
	unsigned long checksum_matches = 0;
	unsigned long checksum_found = 0;

	// Calcola i checksum solo sul processo master (rank 0)
	if (rank == 0) {
		// Somma le posizioni valide in h_pat_found
		for (int i = 0; i < pat_number; i++) {
			if (h_pat_found[i] != (unsigned long)NOT_FOUND) {
				checksum_found += h_pat_found[i];
			}
		}
		checksum_found %= CHECKSUM_MAX; 

		// Somma i valori validi in h_seq_matches
		for (unsigned long i = 0; i < seq_length; i++) {
			if (h_seq_matches[i] != NOT_FOUND) {
				checksum_matches += h_seq_matches[i];
			}
		}
		checksum_matches %= CHECKSUM_MAX; 

		free(h_seq_matches);	// Libera la memoria di h_seq_matches
	}
	free(h_pat_found); // Libera la memoria di h_pat_found (tutti i processi)


#ifdef DEBUG
	/* DEBUG: Write results */
	printf("-----------------\n");
	printf("Found start:");
	for( debug_pat=0; debug_pat<pat_number; debug_pat++ ) {
		printf( " %lu", pat_found[debug_pat] );
	}
	printf("\n");
	printf("-----------------\n");
	printf("Matches:");
	for( lind=0; lind<seq_length; lind++ ) 
		printf( " %d", seq_matches[lind] );
	printf("\n");
	printf("-----------------\n");
#endif // DEBUG

	/* Free local resources */	
	free( sequence );
	free( seq_matches );

/*
 *
 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
 *
 */

	/* 8. Stop global timer */
    CUDA_CHECK_FUNCTION( hipDeviceSynchronize() );
	MPI_Barrier( MPI_COMM_WORLD );
	ttotal = cp_Wtime() - ttotal;

	if (rank == 0) {
		/* 9. Output for leaderboard */
		printf("\n");
		/* 9.1. Total computation time */
		printf("Time: %lf\n", ttotal );

		/* 9.2. Results: Statistics */
		printf("Result: %d, %lu, %lu\n\n", 
				pat_matches,
				checksum_found,
				checksum_matches );
	}
		
	/* 10. Free resources */	
	int i;
	for( i=0; i<pat_number; i++ ) free( pattern[i] );
	free( pattern );
	free( pat_length );
	free( pat_found );


	/* 11. End */
	MPI_Finalize();
	return 0;
}	
